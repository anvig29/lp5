#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
using namespace std;

#define N 10000

__global__ void vectorAdd(int *d_a, int *d_b, int *d_c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        d_c[i] = d_a[i] + d_b[i];
    }
}

class VectorAddition {
public:
    void performVectorAddition() {
        int *a, *b, *c, *d;
        int *d_a, *d_b, *d_c;
        size_t size = N * sizeof(int);

        a = (int *)malloc(size);
        b = (int *)malloc(size);
        c = (int *)malloc(size);
        d = (int *)malloc(size);

        srand(time(NULL));
        for (int i = 0; i < N; i++) {
            a[i] = rand() % 100;
            b[i] = rand() % 100;
        }

        clock_t start_cpu = clock();
        for (int i = 0; i < N; i++) {
            c[i] = a[i] + b[i];
        }
        clock_t end_cpu = clock();
        double cpu_time = (double)(end_cpu - start_cpu) / CLOCKS_PER_SEC;

        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
        hipEventRecord(stop);

        hipMemcpy(d, d_c, size, hipMemcpyDeviceToHost);
        hipEventSynchronize(stop);
        float gpu_time = 0;
        hipEventElapsedTime(&gpu_time, start, stop);

        bool match = true;
        for (int i = 0; i < N; i++) {
            if (c[i] != d[i]) {
                match = false;
                break;
            }
        }

        printf("CPU Time: %.6f s\n", cpu_time);
        printf("GPU Time: %.6f ms\n", gpu_time);
        printf("Speedup Factor: %.2f\n", (cpu_time) * 1000 / gpu_time);
        printf("Arrays Match(Verification): %s\n", match ? "Yes" : "No");

        free(a); free(b); free(c); free(d);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
};

int main() {
    VectorAddition vectorAdder;
    vectorAdder.performVectorAddition();

    //nvcc -version
    //nvidia smi
    //nvcc matMul.cu
    // ./a.out
    return 0;
}
